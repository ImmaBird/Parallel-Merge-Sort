#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "sort.cuh"

#define SIZE 100000000
#define STARTRANGE 0
#define ENDRANGE 10000

#define THREADS_PER_BLOCK 256
#define CHUNK_SIZE 16

// flag if the prng has been seeded
int randNotSeeded = 1;

// tests the gpu merge sort
int main()
{
    // variables to time the sort
    clock_t start, stop;

    // the array to test our sort on
    int *data = getRandomArray(SIZE);

    // print the first 15 elements of the data
    if (SIZE > 15)
    {
        printArray(data, 15);
    }
    else
    {
        printArray(data, SIZE);
    }

    // gets the right answer to compare too at the end
    int *data_qsort = (int*)malloc(SIZE*sizeof(int));
    memcpy(data_qsort, data, SIZE*sizeof(int));

    start = clock();
    qsort(data_qsort, SIZE, sizeof(int), comparator);
    stop = clock();
    double qsort_time = ((double) (stop - start)) / CLOCKS_PER_SEC;
    

    // runs the program and times it
    start = clock();
    mergeSort(data, SIZE);
    stop = clock();
    

    // print the first 15 elements of the hopefully sorted data array
    printf("\n");
    if (SIZE > 15)
    {
        printArray(data_qsort, 15);
    }
    else
    {
        printArray(data_qsort, SIZE);
    }

    // prints the first 15 elements of the sorted array
    if (SIZE > 15)
    {
        printArray(data, 15);
    }
    else
    {
        printArray(data, SIZE);
    }
    compareArrays(data, data_qsort, SIZE);

    // print elapsed time
    double elapsed = ((double) (stop - start)) / CLOCKS_PER_SEC;
    printf("Elapsed time: %.3fs\n", elapsed);
    printf("qsort time: %.3fs\n", qsort_time);

    // Cleanup
    free(data);
    free(data_qsort);
    return 0;
}

// parallel merge sort using a GPU
void mergeSort(int *h_array, int arraySize)
{
    hipError_t err;
    // Make array in gpu memory
    int *d_array;
    hipMalloc((void **)&d_array, arraySize * sizeof(int));
    hipMemcpy(d_array, h_array, arraySize * sizeof(int), hipMemcpyHostToDevice);

    // sort
    int chunkSize = CHUNK_SIZE;
    int chunks = arraySize / chunkSize + 1;
    int blocks = chunks / THREADS_PER_BLOCK + 1;
    gpu_sort<<<blocks, THREADS_PER_BLOCK>>>(d_array, arraySize, chunkSize);
    hipDeviceSynchronize();
    //hipMemcpy(h_array, d_array, arraySize*sizeof(int), hipMemcpyDeviceToHost);
    //hipFree(d_array);

    // merge
    //cpuMerge(h_array, arraySize, chunkSize);
    

    // Make temp array for the merge
    int* d_temp_data;
    hipMalloc((void **)&d_temp_data, arraySize*sizeof(int));
    do
    {
        chunkSize *= 2;
        chunks = arraySize / chunkSize + 1;
        blocks = chunks / THREADS_PER_BLOCK + 1;
        if (chunkSize >= arraySize / 2048)
        {
            hipMemcpy(h_array, d_array, arraySize*sizeof(int), hipMemcpyDeviceToHost);
            hipFree(d_array);
            cpuMerge(h_array, arraySize, chunkSize);
            break;
        }
        gpu_merge<<<blocks, THREADS_PER_BLOCK>>>(d_array, d_temp_data, arraySize, chunkSize);
        err = hipDeviceSynchronize();
        printf("Merge: %s chunkSize: %d\n", hipGetErrorString(err), chunkSize);
    }
    while(chunkSize < arraySize);
}

// sorts a bunch of small chunks from one big array
__global__ void gpu_sort(int *d_array, int size, int chunkSize)
{
    // Figure out left and right for this thread
    int a = (threadIdx.x + blockDim.x * blockIdx.x) * chunkSize;
    if (a >= size) return;

    int b = a + chunkSize;
    if (b > size) b = size;

    insertionSort(d_array, a, b);
}

// merges small sorted arrays into on big one
__global__ void gpu_merge(int *d_array, int *d_temp_array, int arraySize, int chunkSize)
{
    int pos = (threadIdx.x + blockDim.x * blockIdx.x);
    int a = pos * chunkSize;
    if (a >= arraySize) return;
    int halfChunk = chunkSize / 2;
    int m = a + halfChunk;
    if (m >= arraySize) return;
    int b = m + halfChunk;
    if (b > arraySize) b = arraySize;

    mergeArrays(d_array, d_temp_array, a, m, b);

    for (int i = a; i < b; i++)
    {
        d_array[i] = d_temp_array[i];
    }
    //memcpy(d_array+a, d_temp_array+a, (b-a)*sizeof(int));
}

// serial cpu merge chunk size is the size of one sorted arrays
void cpuMerge(int *array, int size, int chunkSize)
{
    int *buffer = (int*)malloc(size*sizeof(int));
    int *data = (int*)malloc(size*sizeof(int));
    memcpy(data, array, size * sizeof(int));
    int *temp;
    int a, b, m, halfChunk;
    
    do
    {
        chunkSize *= 2;
        for (a = 0; a < size; a += chunkSize)
        {
            halfChunk = chunkSize / 2;
            m = a + halfChunk;
            if (m >= size)
            {
                memcpy(buffer+a, data+a, (size - a) * sizeof(int));
                break;
            }
            b = m + halfChunk;
            if (b > size) b = size;

            mergeArrays(data, buffer, a, m, b);
        }

        temp = buffer;
        buffer = data;
        data = temp;
    }
    while (chunkSize < size);

    memcpy(array, data, size * sizeof(int));
    free(buffer);
    free(data);
}

__host__ __device__ void mergeArrays(int *data, int *buffer, int a, int m, int b)
{
    int l, r, i;
    l = a;
    r = m;
    for (i = a; i < b; i++)
    {
        if (data[l] < data[r])
        {
            buffer[i] = data[l];
            l++;
            if (l == m)
            {
                while (r < b)
                {
                    i++;
                    buffer[i] = data[r];
                    r++;
                }
                break;
            }
        }
        else
        {
            buffer[i] = data[r];
            r++;
            if (r == b)
            {
                while (l < m)
                {
                    i++;
                    buffer[i] = data[l];
                    l++;
                }
                break;
            }
        }
    }
}

// sorts an array from [a,b)
__device__ void insertionSort(int *array, int a, int b)
{
    int current;
    for (int i = a + 1; i < b; i++)
    {
        current = array[i];
        for (int j = i - 1; j >= a - 1; j--)
        {
            if (j == a - 1 || current > array[j])
            {
                array[j + 1] = current;
                break;
            }
            else
            {
                array[j + 1] = array[j];
            }
        }
    }
}

// prints an array
__host__ __device__ void printArray(int *d_array, int size)
{
    for (int i = 0; i < size; i++)
    {
        printf("%d ", d_array[i]);
    }
    printf("\n");
}

// gets an array filled with random values
int *getRandomArray(int size)
{
    // seed the prng if needed
    if (randNotSeeded)
    {
        srand(time(0));
        randNotSeeded = 0;
    }

    int *array = (int *)malloc(size * sizeof(int));
    for (int i = 0; i < size; i++)
    {
        array[i] = randInt(STARTRANGE, ENDRANGE);
    }
    return array;
}

// gets a random int in range [a,b)
int randInt(int a, int b)
{
    return (rand() % b) + a;
}

// used by qsort for comparisons
int comparator(const void *p, const void *q)
{
    return *(const int *)p - *(const int *)q;
}

// returns true if success
int compareArrays(int *array1, int *array2, int size)
{
    for (int i = 0; i < size; i++) {
        if (array1[i] != array2[i]) {
            printf("Broken at index:%d :(\n", i);
            return false;
        }
    }
    return true;
}