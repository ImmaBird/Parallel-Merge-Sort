#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "sort.cuh"

#define SIZE 524288
#define THREADS_PER_BLOCK 512

#define STARTRANGE 0
#define ENDRANGE 100

int main()
{
    // TODO: Breaks if using any array size that isn't a power of 2
    // TODO: Also breaks if using an array size that is a power of 2 >= 2^20 :)
    // TODO: Make the above two todos not happen
    srand(time(0));

    int *data;

    data = getRandomArray(SIZE);
    //printArray(data, SIZE);

    clock_t start, stop;
    start = clock();
    mergeSort(data, SIZE);
    stop = clock();

    printf("\n\n");
    double elapsed = ((double) (stop - start)) / CLOCKS_PER_SEC;
    printf("Elapsed time: %.3fs\n", elapsed);

    //printf("\nResult: ");
    //printArray(data, SIZE);

    for (int i = 1; i < SIZE; i++) {
        if (data[i] < data[i-1]) {
            printf("Broken :(\n");
            break;
        }
    }

    // Cleanup
    free(data);

    return 0;
}

void mergeSort(int *array, int arraySize)
{
    // Make array in gpu memory
    int *d_array;
    hipMalloc((void **)&d_array, arraySize*sizeof(int));
    hipMemcpy(d_array, array, arraySize*sizeof(int), hipMemcpyHostToDevice);

    int chunkSize = 2;
    int chunks = arraySize / chunkSize;
    
    int blocks = chunks / THREADS_PER_BLOCK + 1;

    gpu_mergeSort<<<blocks, THREADS_PER_BLOCK>>>(d_array, arraySize, chunkSize);
    hipDeviceSynchronize();

    // Make temp array for the merge
    int* d_temp_data;
    hipMalloc((void **)&d_temp_data, arraySize);
    chunkSize *= 2;
    while(chunkSize <= arraySize)
    {
        gpu_merge<<<blocks, THREADS_PER_BLOCK>>>(d_array, d_temp_data, arraySize, chunkSize);
        // TODO: Make chunkSize math not terrible
        if (chunkSize == arraySize) break;
        chunkSize *= 2;
        if (chunkSize > arraySize) chunkSize = arraySize;
        chunks = arraySize / chunkSize;
        blocks = chunks / THREADS_PER_BLOCK + 1;
        hipDeviceSynchronize();
    }

    // Copy result back to host
    hipMemcpy(array, d_array, arraySize*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_array);
}

__global__ void gpu_mergeSort(int *d_array, int arraySize, int chunkSize)
{
    // Figure out left and right for this thread
    int l = (threadIdx.x + blockDim.x * blockIdx.x) * chunkSize;
    if (l >= arraySize) return;
    int r = l + chunkSize;
    if (r > arraySize) r = arraySize;

    insertionSort(d_array, l, r);
}

__global__ void gpu_merge(int *d_array, int *d_temp_array, int arraySize, int chunkSize)
{
    // Figure out left and right for this thread
    int l = (threadIdx.x + blockDim.x * blockIdx.x) * chunkSize;
    if (l >= arraySize) return;
    int r = l + chunkSize;
    if (r > arraySize) r = arraySize;

    int cur_l = l;
    int m = (r - l) / 2 + l;
    int cur_r = m;

    // if (threadIdx.x == 0)
    // {
    //     printf("(Before) Block: %d l: %d r: %d m: %d\n", blockIdx.x, l, r, m);
    //     for (int i = l; i < r; i++)
    //     {
    //         printf("%d ", d_array[i]);
    //     }
    //     printf("\n");
    // }

    for (int i = l; i < r; i++)
    {
        if (cur_r >= r || (cur_l < m && d_array[cur_l] < d_array[cur_r]))
        {
            // left less than right
            d_temp_array[i] = d_array[cur_l];
            cur_l++;
        }
        else
        {
            // right less than left
            d_temp_array[i] = d_array[cur_r];
            cur_r++;
        }     
    }

    memcpy(d_array+l, d_temp_array+l, chunkSize*sizeof(int));

    // if (threadIdx.x == 0)
    // {
    //     printf("(After) Block: %d l: %d r: %d\n", blockIdx.x, l, r);
    //     for (int i = l; i < r; i++)
    //     {
    //         printf("%d ", d_array[i]);
    //     }
    //     printf("\n");
    // }
}

__device__ void insertionSort(int *array, int a, int b)
{
    int current;
    for (int i = a + 1; i < b; i++)
    {
        current = array[i];
        for (int j = i - 1; j >= a - 1; j--)
        {
            if (j == a - 1 || current > array[j])
            {
                array[j + 1] = current;
                break;
            }
            else
            {
                array[j + 1] = array[j];
            }
        }
    }
}

__device__ void selectionSort(int *array, int size)
{
    for (int i = 0; i < size; i++)
    {
        int smallest = array[i];
        int index = i;
        for (int j = i + 1; j < size; j++)
        {
            if (array[j] < smallest)
            {
                smallest = array[j];
                index = j;
            }
        }
        array[index] = array[i];
        array[i] = smallest;
    }
}

void printArray(int *array, int size)
{
    for (int i = 0; i < size; i++)
    {
        printf("%d ", array[i]);
    }
    printf("\n");
}

__global__ void printGPUArray(int *d_array, int size)
{
    for (int i = 0; i < size; i++)
    {
        printf("%d ", d_array[i]);
    }
    printf("\n");
}

int *getRandomArray(int size)
{
    int *array = (int *)malloc(size * sizeof(int));
    for (int i = 0; i < size; i++)
    {
        array[i] = randInt(STARTRANGE, ENDRANGE);
    }
    return array;
}

int randInt(int a, int b)
{
    return (rand() % b) + a;
}
