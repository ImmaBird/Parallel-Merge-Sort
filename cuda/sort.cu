#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "sort.cuh"

#define THREADS_PER_BLOCK 256
#define CHUNK_SIZE 16

// flag if the prng has been seeded
int randNotSeeded = 1;

// tests the gpu merge sort
int main(int argc, char** argv)
{
    // variables to time the sort
    double start, stop;

    // Get cmdline args
    if (argc != 4)
    {
        printf("USAGE: %s arrayLength minValue maxValue\n", argv[0]);
        return 1;
    }

    int arrayLength = atoi(argv[1]);
    int minValue = atoi(argv[2]);
    int maxValue = atoi(argv[3]);

    // the array to test our sort on
    int *data = getRandomArray(arrayLength, minValue, maxValue);

    // gets the right answer to compare too at the end
    // int *data_qsort = (int*)malloc(arrayLength*sizeof(int));
    // memcpy(data_qsort, data, arrayLength*sizeof(int));

    // Run quick sort to have an array to check against for validation
    // start = omp_get_wtime();
    // qsort(data_qsort, arrayLength, sizeof(int), comparator);
    // stop = omp_get_wtime();
    // double qsort_time = stop - start;
    

    // runs the program and times it
    start = clock();
    mergeSort(data, arrayLength);
    stop = clock();

    // Validate
    // compareArrays(data, data_qsort, arrayLength);

    // print elapsed time
    double elapsed = (stop - start) / CLOCKS_PER_SEC;
    printf("%d, %.5f\n", arrayLength, elapsed);
    // printf("qsort time: %.3fs\n", qsort_time);

    // Cleanup
    free(data);
    // free(data_qsort);
    return 0;
}

// parallel merge sort using a GPU
void mergeSort(int *h_array, int arraySize)
{
    // Make array in gpu memory
    int *d_array;
    hipMalloc((void **)&d_array, arraySize * sizeof(int));
    hipMemcpy(d_array, h_array, arraySize * sizeof(int), hipMemcpyHostToDevice);

    // sort
    int chunkSize = CHUNK_SIZE;
    int chunks = arraySize / chunkSize + 1;
    int blocks = chunks / THREADS_PER_BLOCK + 1;
    gpu_sort<<<blocks, THREADS_PER_BLOCK>>>(d_array, arraySize, chunkSize);
    hipDeviceSynchronize();

    // Make temp array for the merge
    int* d_temp_data;
    hipMalloc((void **)&d_temp_data, arraySize*sizeof(int));
    do
    {
        chunkSize *= 2;
        chunks = arraySize / chunkSize + 1;
        blocks = chunks / THREADS_PER_BLOCK + 1;
        if (blocks < 8)
        {
            // CPU does the merges
            hipMemcpy(h_array, d_array, arraySize*sizeof(int), hipMemcpyDeviceToHost);
            cpuMerge(h_array, arraySize, chunkSize/2);
            break;
        }

        // GPU does the merges
        gpu_merge<<<blocks, THREADS_PER_BLOCK>>>(d_array, d_temp_data, arraySize, chunkSize);
        hipDeviceSynchronize();
    }
    while(chunkSize <= arraySize);
    
    // Free GPU memory
    hipFree(d_array);
    hipFree(d_temp_data);
}

// sorts a bunch of small chunks from one big array
__global__ void gpu_sort(int *d_array, int size, int chunkSize)
{
    // Figure out left and right for this thread
    int a = (threadIdx.x + blockDim.x * blockIdx.x) * chunkSize;
    if (a >= size) return;

    int b = a + chunkSize;
    if (b > size) b = size;

    insertionSort(d_array, a, b);
}

// merges small sorted arrays into on big one
__global__ void gpu_merge(int *d_array, int *d_temp_array, int arraySize, int chunkSize)
{
    int pos = (threadIdx.x + blockDim.x * blockIdx.x);
    int a = pos * chunkSize;
    if (a >= arraySize) return;
    int halfChunk = chunkSize / 2;
    int m = a + halfChunk;
    if (m >= arraySize) return;
    int b = m + halfChunk;
    if (b > arraySize) b = arraySize;

    // Watch out for integer overflow
    if (a < 0 || m < 0 || b < 0) return;

    mergeArrays(d_array, d_temp_array, a, m, b);

    memcpy(d_array+a, d_temp_array+a, (b-a)*sizeof(int));
}

// serial cpu merge chunk size is the size of one sorted arrays
void cpuMerge(int *array, int size, int chunkSize)
{
    int *buffer = (int*)malloc(size*sizeof(int));
    int *data = (int*)malloc(size*sizeof(int));
    memcpy(data, array, size * sizeof(int));
    int *temp;
    int a, b, m, halfChunk;
    
    do
    {
        chunkSize *= 2;

        halfChunk = chunkSize / 2;
        for (a = 0; a < size; a += chunkSize)
        {
            m = a + halfChunk;
            if (m >= size)
            {
                memcpy(buffer+a, data+a, (size - a) * sizeof(int));
                break;
            }
            b = m + halfChunk;
            if (b > size) b = size;

            mergeArrays(data, buffer, a, m, b);
        }

        temp = buffer;
        buffer = data;
        data = temp;
    }
    while (chunkSize < size);

    memcpy(array, data, size * sizeof(int));
    free(data);
    free(buffer);
}

// Merge two sides of an array [a, m) and [m, b)
__host__ __device__ void mergeArrays(int *data, int *buffer, int a, int m, int b)
{
    int l, r, i;
    l = a;
    r = m;
    for (i = a; i < b; i++)
    {
        if (data[l] < data[r])
        {
            buffer[i] = data[l];
            l++;
            if (l == m)
            {
                while (r < b)
                {
                    i++;
                    buffer[i] = data[r];
                    r++;
                }
                break;
            }
        }
        else
        {
            buffer[i] = data[r];
            r++;
            if (r == b)
            {
                while (l < m)
                {
                    i++;
                    buffer[i] = data[l];
                    l++;
                }
                break;
            }
        }
    }
}

// sorts an array from [a,b)
__device__ void insertionSort(int *array, int a, int b)
{
    int current;
    for (int i = a + 1; i < b; i++)
    {
        current = array[i];
        for (int j = i - 1; j >= a - 1; j--)
        {
            if (j == a - 1 || current > array[j])
            {
                array[j + 1] = current;
                break;
            }
            else
            {
                array[j + 1] = array[j];
            }
        }
    }
}

// prints an array
__host__ __device__ void printArray(int *d_array, int size)
{
    for (int i = 0; i < size; i++)
    {
        printf("%d ", d_array[i]);
    }
    printf("\n");
}

// gets an array filled with random values
int *getRandomArray(int size, int startRange, int endRange)
{
    // seed the prng if needed
    if (randNotSeeded)
    {
        srand(time(0));
        randNotSeeded = 0;
    }

    int *array = (int *)malloc(size * sizeof(int));
    for (int i = 0; i < size; i++)
    {
        array[i] = randInt(startRange, endRange);
    }
    return array;
}

// gets a random int in range [a,b)
int randInt(int a, int b)
{
    return (rand() % b) + a;
}

// used by qsort for comparisons
int comparator(const void *p, const void *q)
{
    return *(const int *)p - *(const int *)q;
}

// returns true if success
int compareArrays(int *array1, int *array2, int size)
{
    for (int i = 0; i < size; i++) {
        if (array1[i] != array2[i]) {
            printf("Broken at index:%d :(\n", i);
            return false;
        }
    }
    return true;
}
